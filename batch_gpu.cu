
#include "nbd.hxx"
#include "profile.hxx"

#include "hip/hip_runtime_api.h"
#include "hipblas.h"
#include "hipsolver.h"
#include "hiprand.h"

#include <vector>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/iterator/permutation_iterator.h>

#include <cassert>
#include <cstdlib>
#include <cstring>

extern hipStream_t stream;
extern hipblasHandle_t cublasH;
extern hipsolverHandle_t cusolverH;

void set_work_size(int64_t Lwork, double** D_DATA, int64_t* D_DATA_SIZE) {
  if (Lwork > *D_DATA_SIZE) {
    *D_DATA_SIZE = Lwork;
    if (*D_DATA)
      hipFree(*D_DATA);
    hipMalloc((void**)D_DATA, sizeof(double) * Lwork);
  }
  else if (Lwork <= 0) {
    *D_DATA_SIZE = 0;
    if (*D_DATA)
      hipFree(*D_DATA);
  }
}

void allocBufferedList(void** A_ptr, void** A_buffer, int64_t element_size, int64_t count) {
  int64_t bytes = element_size * count;
  hipMalloc((void**)A_ptr, bytes);
  *A_buffer = malloc(bytes);
  memset((void*)*A_buffer, 0, bytes);
}

void flushBuffer(char dir, void* A_ptr, void* A_buffer, int64_t element_size, int64_t count) {
  int64_t bytes = element_size * count;
  if (dir == 'G' || dir == 'g')
    hipMemcpy(A_buffer, A_ptr, bytes, hipMemcpyDeviceToHost);
  else if (dir == 'S' || dir == 's')
    hipMemcpy(A_ptr, A_buffer, bytes, hipMemcpyHostToDevice);
}

void freeBufferedList(void* A_ptr, void* A_buffer) {
  hipFree(A_ptr);
  free(A_buffer);
}

struct BatchedFactorParams {
  int64_t N_r, N_s, N_upper, L_diag, L_nnz, L_tmp;
  const double** A_d, **U_d, **U_r, **U_s, **V_x, **A_rs, **A_sx, *U_d0;
  double** U_dx, **A_x, **B_x, **A_ss, **A_upper, *UD_data, *A_data, *B_data;
  int* info;
};

void batchParamsCreate(void** params, int64_t R_dim, int64_t S_dim, const double* U_ptr, double* A_ptr, double* X_ptr, int64_t N_up, double** A_up, double** X_up,
  double* Workspace, int64_t Lwork, int64_t N_rows, int64_t N_cols, int64_t col_offset, const int64_t row_A[], const int64_t col_A[]) {
  
  int64_t N_dim = R_dim + S_dim;
  int64_t NNZ = col_A[N_cols] - col_A[0];
  int64_t stride = N_dim * N_dim;
  int64_t lenB = (Lwork / stride) - N_cols;
  lenB = lenB > NNZ ? NNZ : lenB;

  const double** _A_d, **_U_d, **_U_r, **_U_s, **_V_x, **_A_rs, **_A_sx;
  double** _U_dx, **_A_x, **_B_x, **_A_ss, **_A_upper;
  _A_d = (const double**)malloc(sizeof(double*) * N_cols);
  _U_d = (const double**)malloc(sizeof(double*) * N_cols);
  _U_r = (const double**)malloc(sizeof(double*) * NNZ);
  _U_s = (const double**)malloc(sizeof(double*) * NNZ);
  _V_x = (const double**)malloc(sizeof(double*) * NNZ);
  _A_rs = (const double**)malloc(sizeof(double*) * N_cols);
  _A_sx = (const double**)malloc(sizeof(double*) * lenB);

  _U_dx = (double**)malloc(sizeof(double*) * N_cols);
  _A_x = (double**)malloc(sizeof(double*) * NNZ);
  _B_x = (double**)malloc(sizeof(double*) * lenB);
  _A_ss = (double**)malloc(sizeof(double*) * N_cols);
  _A_upper = (double**)malloc(sizeof(double*) * NNZ);

  double* _UD_data = Workspace;
  double* _B_data = &Workspace[N_cols * stride];
  const double* _U_d0 = U_ptr + stride * col_offset;

  for (int64_t x = 0; x < N_cols; x++) {
    int64_t diag_id = 0;
    for (int64_t yx = col_A[x]; yx < col_A[x + 1]; yx++) {
      int64_t y = row_A[yx];
      if (x + col_offset == y)
        diag_id = yx;
      _U_r[yx] = U_ptr + stride * y;
      _U_s[yx] = U_ptr + stride * y + R_dim * N_dim;
      _V_x[yx] = _UD_data + stride * x;
      _A_x[yx] = A_ptr + stride * yx;
      _A_upper[yx] = A_up[yx];
    }

    _A_d[x] = A_ptr + stride * diag_id;
    _U_d[x] = U_ptr + stride * (x + col_offset);
    _A_rs[x] = A_ptr + stride * diag_id + R_dim;
    _U_dx[x] = _UD_data + stride * x;
    _A_ss[x] = A_up[diag_id];
  }

  for (int64_t x = 0; x < lenB; x++) {
    _B_x[x] = _B_data + stride * x;
    _A_sx[x] = _B_data + stride * x + R_dim * N_dim;
  }
  
  struct BatchedFactorParams* params_ptr = (struct BatchedFactorParams*)malloc(sizeof(struct BatchedFactorParams));
  params_ptr->N_r = R_dim;
  params_ptr->N_s = S_dim;
  params_ptr->N_upper = N_up;
  params_ptr->L_diag = N_cols;
  params_ptr->L_nnz = NNZ;
  params_ptr->L_tmp = lenB;

  hipMalloc((void**)&(params_ptr->A_d), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->U_d), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->U_r), sizeof(double*) * NNZ);
  hipMalloc((void**)&(params_ptr->U_s), sizeof(double*) * NNZ);
  hipMalloc((void**)&(params_ptr->V_x), sizeof(double*) * NNZ);
  hipMalloc((void**)&(params_ptr->A_rs), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->A_sx), sizeof(double*) * lenB);

  hipMalloc((void**)&(params_ptr->U_dx), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->A_x), sizeof(double*) * NNZ);
  hipMalloc((void**)&(params_ptr->B_x), sizeof(double*) * lenB);
  hipMalloc((void**)&(params_ptr->A_ss), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->A_upper), sizeof(double*) * NNZ);

  params_ptr->U_d0 = _U_d0;
  params_ptr->UD_data = _UD_data;
  params_ptr->A_data = A_ptr;
  params_ptr->B_data = _B_data;

  hipMalloc((void**)&(params_ptr->info), sizeof(int) * N_cols);
  *params = params_ptr;

  hipMemcpy(params_ptr->A_d, _A_d, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->U_d, _U_d, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->U_r, _U_r, sizeof(double*) * NNZ, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->U_s, _U_s, sizeof(double*) * NNZ, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->V_x, _V_x, sizeof(double*) * NNZ, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_rs, _A_rs, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_sx, _A_sx, sizeof(double*) * lenB, hipMemcpyHostToDevice);

  hipMemcpy(params_ptr->U_dx, _U_dx, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_x, _A_x, sizeof(double*) * NNZ, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->B_x, _B_x, sizeof(double*) * lenB, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_ss, _A_ss, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_upper, _A_upper, sizeof(double*) * NNZ, hipMemcpyHostToDevice);

  free(_A_d);
  free(_U_d);
  free(_U_r);
  free(_U_s);
  free(_V_x);
  free(_A_rs);
  free(_A_sx);
  free(_U_dx);
  free(_A_x);
  free(_B_x);
  free(_A_ss);
  free(_A_upper);
}

void batchParamsDestory(void* params) {
  struct BatchedFactorParams* params_ptr = (struct BatchedFactorParams*)params;
  if (params_ptr->A_d)
    hipFree(params_ptr->A_d);
  if (params_ptr->U_d)
    hipFree(params_ptr->U_d);
  if (params_ptr->U_r)
    hipFree(params_ptr->U_r);
  if (params_ptr->U_s)
    hipFree(params_ptr->U_s);
  if (params_ptr->V_x)
    hipFree(params_ptr->V_x);
  if (params_ptr->A_rs)
    hipFree(params_ptr->A_rs);
  if (params_ptr->A_sx)
    hipFree(params_ptr->A_sx);
  if (params_ptr->U_dx)
    hipFree(params_ptr->U_dx);
  if (params_ptr->A_x)
    hipFree(params_ptr->A_x);
  if (params_ptr->B_x)
    hipFree(params_ptr->B_x);
  if (params_ptr->A_ss)
    hipFree(params_ptr->A_ss);
  if (params_ptr->A_upper)
    hipFree(params_ptr->A_upper);
  if (params_ptr->info)
    hipFree(params_ptr->info);

  free(params);
}

void batchCholeskyFactor(void* params_ptr, const struct CellComm* comm) {
  struct BatchedFactorParams* params = (struct BatchedFactorParams*)params_ptr;
  int64_t U = params->N_upper, R = params->N_r, S = params->N_s, N = R + S, D = params->L_diag;
  int64_t alen = N * N * params->L_nnz;
  double one = 1., zero = 0., minus_one = -1.;

#ifdef _PROF
  double stime = MPI_Wtime();
#endif
  level_merge_gpu(params->A_data, alen, stream, comm);
  dup_bcast_gpu(params->A_data, alen, stream, comm);
#ifdef _PROF
  hipStreamSynchronize(stream);
  double etime = MPI_Wtime() - stime;
  recordCommTime(etime);
#endif

  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, R, N, &one, 
    params->A_d, N, params->U_d, N, &zero, params->U_dx, N, D);
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, R, R, N, &one, 
    params->U_d, N, (const double**)(params->U_dx), N, &zero, params->B_x, N, D);
  hipblasDcopy(cublasH, N * N * D, params->U_d0, 1, params->UD_data, 1);

  hipsolverDnDpotrfBatched(cusolverH, HIPBLAS_FILL_MODE_LOWER, R, params->B_x, N, params->info, D);
  hipblasDtrsmBatched(cublasH, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, 
    N, R, &one, (const double**)(params->B_x), N, params->U_dx, N, D);

  for (int64_t i = 0; i < params->L_nnz; i += params->L_tmp) {
    int64_t len = params->L_nnz - i > params->L_tmp ? params->L_tmp : params->L_nnz - i;
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &one, 
      (const double**)(&params->A_x[i]), N, &params->V_x[i], N, &zero, params->B_x, N, len);
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, N, R, N, &one, 
      &params->U_r[i], N, (const double**)(params->B_x), N, &zero, &params->A_x[i], N, len);
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, S, S, N, &one, 
      &params->U_s[i], N, params->A_sx, N, &zero, &params->A_upper[i], U, len);
  }
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, S, S, R, &minus_one, 
    params->A_rs, N, params->A_rs, N, &one, params->A_ss, U, D);
  
  hipStreamSynchronize(stream);
}

struct LastFactorParams {
  double *A_ptr, *Workspace;
  int64_t N_A;
  int Lwork, *info;
};

void lastParamsCreate(void** params, double* A, int64_t N) {
  struct LastFactorParams* params_ptr = (struct LastFactorParams*)malloc(sizeof(struct LastFactorParams));
  *params = params_ptr;

  params_ptr->A_ptr = A;
  params_ptr->N_A = N;

  hipsolverDnDpotrf_bufferSize(cusolverH, HIPBLAS_FILL_MODE_LOWER, N, A, N, &params_ptr->Lwork);
  hipMalloc((void**)&params_ptr->Workspace, sizeof(double) * params_ptr->Lwork);
  hipMalloc((void**)&params_ptr->info, sizeof(int));
}

void lastParamsDestory(void* params) {
  struct LastFactorParams* params_ptr = (struct LastFactorParams*)params;
  if (params_ptr->Workspace)
    hipFree(params_ptr->Workspace);
  if (params_ptr->info)
    hipFree(params_ptr->info);
  
  free(params);
}

void chol_decomp(void* params_ptr, const struct CellComm* comm) {
  struct LastFactorParams* params = (struct LastFactorParams*)params_ptr;
  double* A = params->A_ptr;
  int64_t N = params->N_A;
  int64_t alen = N * N;

#ifdef _PROF
  double stime = MPI_Wtime();
#endif
  level_merge_gpu(params->A_ptr, alen, stream, comm);
  dup_bcast_gpu(params->A_ptr, alen, stream, comm);
#ifdef _PROF
  hipStreamSynchronize(stream);
  double etime = MPI_Wtime() - stime;
  recordCommTime(etime);
#endif

  hipsolverDnDpotrf(cusolverH, HIPBLAS_FILL_MODE_LOWER, N, A, N, params->Workspace, params->Lwork, params->info);
  hipStreamSynchronize(stream);
}

