#include "hip/hip_runtime.h"

#include "nbd.h"
#include "hip/hip_runtime_api.h"
#include "hipblas.h"
#include "hipsolver.h"

#include <stdlib.h>

hipStream_t stream = NULL;
hipblasHandle_t cublasH = NULL;
hipsolverHandle_t cusolverH = NULL;

void init_batch_lib() {
  int mpi_rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  int num_device;
  hipGetDeviceCount(&num_device);
  int device = mpi_rank % num_device;
  hipSetDevice(device);

  hipStreamCreate(&stream);
  hipblasCreate(&cublasH);
  hipblasSetStream(cublasH, stream);

  hipsolverDnCreate(&cusolverH);
  hipsolverSetStream(cusolverH, stream);
}

void finalize_batch_lib() {
  if (stream)
    hipStreamDestroy(stream);
  if (cublasH)
    hipblasDestroy(cublasH);
  if (cusolverH)
    hipsolverDnDestroy(cusolverH);
}

void sync_batch_lib() {
  hipStreamSynchronize(stream);
}

void alloc_matrices_aligned(double** A_ptr, int64_t M, int64_t N, int64_t count) {
  int64_t stride = M * N;
  int64_t bytes = sizeof(double) * count * stride;
  hipMalloc((void**)A_ptr, bytes);
  hipMemset(*A_ptr, 0, bytes);
}

void free_matrices(double* A_ptr) {
  hipFree(A_ptr);
}

void copy_basis(char dir, const double* Ur_in, const double* Us_in, double* U_out, int64_t IR_dim, int64_t IS_dim, int64_t OR_dim, int64_t OS_dim, int64_t ldu_in, int64_t ldu_out) {
  IR_dim = IR_dim < OR_dim ? IR_dim : OR_dim;
  IS_dim = IS_dim < OS_dim ? IS_dim : OS_dim;
  int64_t N_in = IR_dim + IS_dim;

  int64_t width = sizeof(double) * N_in;
  int64_t dpitch = sizeof(double) * ldu_out;
  int64_t spitch = sizeof(double) * ldu_in;

  if (dir == 'G') {
    hipMemcpy2DAsync(U_out, dpitch, Ur_in, spitch, width, IR_dim, hipMemcpyDeviceToHost, stream);
    hipMemcpy2DAsync(U_out + OR_dim * ldu_out, dpitch, Us_in, spitch, width, IS_dim, hipMemcpyDeviceToHost, stream);
  }
  else if (dir == 'S') {
    hipMemcpy2DAsync(U_out, dpitch, Ur_in, spitch, width, IR_dim, hipMemcpyHostToDevice, stream);
    hipMemcpy2DAsync(U_out + OR_dim * ldu_out, dpitch, Us_in, spitch, width, IS_dim, hipMemcpyHostToDevice, stream);
  }
}

void copy_mat(char dir, const double* A_in, double* A_out, int64_t M_in, int64_t N_in, int64_t lda_in, int64_t M_out, int64_t N_out, int64_t lda_out) {
  M_in = M_in < M_out ? M_in : M_out;
  N_in = N_in < N_out ? N_in : N_out;

  int64_t width = sizeof(double) * M_in;
  int64_t dpitch = sizeof(double) * lda_out;
  int64_t spitch = sizeof(double) * lda_in;

  if (dir == 'G')
    hipMemcpy2DAsync(A_out, dpitch, A_in, spitch, width, N_in, hipMemcpyDeviceToHost, stream);
  else if (dir == 'S')
    hipMemcpy2DAsync(A_out, dpitch, A_in, spitch, width, N_in, hipMemcpyHostToDevice, stream);
}

__global__ void args_kernel(int64_t R_dim, int64_t S_dim, const double* U_ptr, double* A_ptr, int64_t N_cols, int64_t col_offset, const int64_t row_A[], const int64_t col_A[],
  double* UD_data, double* B_data, const double** A_lis_diag, const double** U_lis_diag, const double** U_lis, const double** V_lis, const double** ARS_lis,
  double** D_lis, double** UD_lis, double** A_lis, double** B_lis, double** ASS_lis) {
  
  int64_t N_dim = R_dim + S_dim;
  int64_t stride = N_dim * N_dim;

  for (int64_t x = blockIdx.x; x < N_cols; x += gridDim.x) {
    for (int64_t yx = col_A[x] + threadIdx.x; yx < col_A[x + 1]; yx += blockDim.x) {
      int64_t y = row_A[yx];
      if (x + col_offset == y) {
        A_lis_diag[x] = A_ptr + stride * yx;
        ARS_lis[x] = A_ptr + stride * yx + R_dim;
        ASS_lis[x] = A_ptr + stride * yx + (N_dim + 1) * R_dim;
      }
  
      U_lis[yx] = U_ptr + stride * y;
      V_lis[yx] = UD_data + stride * x;
      A_lis[yx] = A_ptr + stride * yx;
      B_lis[yx] = B_data + stride * yx;
    }
  
    if (threadIdx.x == 0) {
      U_lis_diag[x] = U_ptr + stride * (x + col_offset);
      D_lis[x] = B_data + stride * x;
      UD_lis[x] = UD_data + stride * x;
    }
  }
}

__global__ void diag_process_kernel(double* D_data, int64_t N_dim, int64_t N) {
  int64_t stride_m = N_dim * N_dim;
  int64_t stride_row = N_dim + 1;
  int64_t rem = N_dim % 8;
  int64_t N_dim_rem = N_dim - rem;

  for (int64_t b = blockIdx.x; b < N; b += gridDim.x) {
    double* data = D_data + stride_m * b;
    for (int64_t i = threadIdx.x * 8; i < N_dim_rem; i += blockDim.x * 8)
      for (int64_t n = 0; n < 8; n++) {
        int64_t loc = (i + n) * stride_row;
        double d = data[loc];
        d = (d == 0.) ? 1. : d;
        data[loc] = d;
      }
  
    if (threadIdx.x < rem) {
      int64_t loc = (N_dim_rem + threadIdx.x) * stride_row; 
      double d = data[loc];
      d = (d == 0.) ? 1. : d;
      data[loc] = d;
    }
  }
}

void batch_cholesky_factor(int64_t R_dim, int64_t S_dim, const double* U_ptr, double* A_ptr, int64_t N_cols, int64_t col_offset, const int64_t row_A[], const int64_t col_A[]) {
  int64_t N_dim = R_dim + S_dim;
  int64_t NNZ = col_A[N_cols] - col_A[0];
  int64_t stride = N_dim * N_dim;

  const double** A_lis_diag, **U_lis_diag, **U_lis, **V_lis, **ARS_lis;
  hipMalloc((void**)&A_lis_diag, sizeof(double*) * N_cols);
  hipMalloc((void**)&U_lis_diag, sizeof(double*) * N_cols);
  hipMalloc((void**)&U_lis, sizeof(double*) * NNZ);
  hipMalloc((void**)&V_lis, sizeof(double*) * NNZ);
  hipMalloc((void**)&ARS_lis, sizeof(double*) * N_cols);

  double** D_lis, **UD_lis, **A_lis, **B_lis, **ASS_lis;
  hipMalloc((void**)&D_lis, sizeof(double*) * N_cols);
  hipMalloc((void**)&UD_lis, sizeof(double*) * N_cols);
  hipMalloc((void**)&A_lis, sizeof(double*) * NNZ);
  hipMalloc((void**)&B_lis, sizeof(double*) * NNZ);
  hipMalloc((void**)&ASS_lis, sizeof(double*) * N_cols);

  double *UD_data, *B_data;
  hipMalloc((void**)&UD_data, sizeof(double) * N_cols * stride);
  hipMalloc((void**)&B_data, sizeof(double) * NNZ * stride);

  int* info_array;
  int64_t *col_arr, *row_arr;
  hipMalloc((void**)&info_array, sizeof(int) * N_cols);
  hipMalloc((void**)&col_arr, sizeof(int64_t) * (N_cols + 1));
  hipMalloc((void**)&row_arr, sizeof(int64_t) * NNZ);

  hipMemcpyAsync((void*)col_arr, (void*)col_A, sizeof(int64_t) * (N_cols + 1), hipMemcpyHostToDevice);
  hipMemcpyAsync((void*)row_arr, (void*)row_A, sizeof(int64_t) * NNZ, hipMemcpyHostToDevice);
  args_kernel<<<8, 256, 0, stream>>>(R_dim, S_dim, U_ptr, A_ptr, N_cols, col_offset, row_arr, col_arr,
    UD_data, B_data, A_lis_diag, U_lis_diag, U_lis, V_lis, ARS_lis, D_lis, UD_lis, A_lis, B_lis, ASS_lis);

  double one = 1., zero = 0., minus_one = -1.;
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N_dim, R_dim, N_dim, &one, 
    A_lis_diag, N_dim, U_lis_diag, N_dim, &zero, UD_lis, N_dim, N_cols);
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, R_dim, R_dim, N_dim, &one, 
    U_lis_diag, N_dim, (const double**)UD_lis, N_dim, &zero, D_lis, N_dim, N_cols);
  hipblasDcopy(cublasH, stride * N_cols, U_ptr + stride * col_offset, 1, UD_data, 1);

  diag_process_kernel<<<8, 256, 0, stream>>>(B_data, N_dim, N_cols);
  hipsolverDnDpotrfBatched(cusolverH, HIPBLAS_FILL_MODE_LOWER, R_dim, D_lis, N_dim, info_array, N_cols);
  hipblasDtrsmBatched(cublasH, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, 
    N_dim, R_dim, &one, D_lis, N_dim, UD_lis, N_dim, N_cols);

  hipblasDgemmBatched(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, N_dim, N_dim, N_dim, &one, 
    U_lis, N_dim, (const double**)A_lis, N_dim, &zero, B_lis, N_dim, NNZ);
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N_dim, N_dim, N_dim, &one, 
    (const double**)B_lis, N_dim, V_lis, N_dim, &zero, A_lis, N_dim, NNZ);
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, S_dim, S_dim, R_dim, &minus_one, 
    ARS_lis, N_dim, ARS_lis, N_dim, &one, ASS_lis, N_dim, N_cols);

  hipFree(A_lis_diag);
  hipFree(U_lis_diag);
  hipFree(U_lis);
  hipFree(V_lis);
  hipFree(ARS_lis);

  hipFree(D_lis);
  hipFree(UD_lis);
  hipFree(A_lis);
  hipFree(B_lis);
  hipFree(ASS_lis);

  hipFree(UD_data);
  hipFree(B_data);
  hipFree(info_array);
  hipFree(col_arr);
  hipFree(row_arr);
}


