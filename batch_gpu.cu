
#include "nbd.h"
#include "hip/hip_runtime_api.h"
#include "hipblas.h"
#include "hipsolver.h"
#include "mkl.h"

#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/iterator/permutation_iterator.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

hipStream_t stream = NULL;
hipblasHandle_t cublasH = NULL;
hipsolverHandle_t cusolverH = NULL;

void init_batch_lib() {
  int mpi_rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  int num_device;
  hipGetDeviceCount(&num_device);
  const char* env = getenv("PROCS_PER_DEVICE");
  int procs_per_device = env == NULL ? 1 : atoi(env);
  int device = (mpi_rank / procs_per_device) % num_device;
  hipSetDevice(device);
  
  hipStreamCreate(&stream);
  hipblasCreate(&cublasH);
  hipblasSetStream(cublasH, stream);

  hipsolverDnCreate(&cusolverH);
  hipsolverSetStream(cusolverH, stream);
}

void finalize_batch_lib() {
  if (stream)
    hipStreamDestroy(stream);
  if (cublasH)
    hipblasDestroy(cublasH);
  if (cusolverH)
    hipsolverDnDestroy(cusolverH);
  stream = NULL;
  cublasH = NULL;
  cusolverH = NULL;
}

void set_work_size(int64_t Lwork, double** D_DATA, int64_t* D_DATA_SIZE) {
  if (Lwork > *D_DATA_SIZE) {
    *D_DATA_SIZE = Lwork;
    if (*D_DATA)
      hipFree(*D_DATA);
    hipMalloc((void**)D_DATA, sizeof(double) * Lwork);
  }
  else if (Lwork <= 0) {
    *D_DATA_SIZE = 0;
    if (*D_DATA)
      hipFree(*D_DATA);
  }
}

void allocBufferedList(void** A_ptr, void** A_buffer, int64_t element_size, int64_t count) {
  int64_t bytes = element_size * count;
  hipMalloc((void**)A_ptr, bytes);
  *A_buffer = malloc(bytes);
  memset((void*)*A_buffer, 0, bytes);
}

void flushBuffer(char dir, void* A_ptr, void* A_buffer, int64_t element_size, int64_t count) {
  int64_t bytes = element_size * count;
  if (dir == 'G' || dir == 'g')
    hipMemcpy(A_buffer, A_ptr, bytes, hipMemcpyDeviceToHost);
  else if (dir == 'S' || dir == 's')
    hipMemcpy(A_ptr, A_buffer, bytes, hipMemcpyHostToDevice);
}

void freeBufferedList(void* A_ptr, void* A_buffer) {
  hipFree(A_ptr);
  free(A_buffer);
}

struct BatchedFactorParams { 
  int64_t N_r, N_s, N_upper, L_diag, L_nnz, L_fill, *F_d;
  const double** A_d, **U_d, **U_r, **U_s, **V_x, **A_rs, **A_sx, *U_d0;
  double** U_dx, **A_x, **B_x, **A_ss, **A_upper, *UD_data, *B_data;
  int* info;
};

void batchParamsCreate(void** params, int64_t R_dim, int64_t S_dim, const double* U_ptr, double* A_ptr, int64_t N_up, double** A_up, double* Workspace,
  int64_t N_cols, int64_t col_offset, const int64_t row_A[], const int64_t col_A[], const int64_t dimr[]) {
  
  int64_t N_dim = R_dim + S_dim;
  int64_t NNZ = col_A[N_cols] - col_A[0];
  int64_t stride = N_dim * N_dim;

  const double** _A_d, **_U_d, **_U_r, **_U_s, **_V_x, **_A_rs, **_A_sx;
  double** _U_dx, **_A_x, **_B_x, **_A_ss, **_A_upper;
  _A_d = (const double**)malloc(sizeof(double*) * N_cols);
  _U_d = (const double**)malloc(sizeof(double*) * N_cols);
  _U_r = (const double**)malloc(sizeof(double*) * NNZ);
  _U_s = (const double**)malloc(sizeof(double*) * NNZ);
  _V_x = (const double**)malloc(sizeof(double*) * NNZ);
  _A_rs = (const double**)malloc(sizeof(double*) * N_cols);
  _A_sx = (const double**)malloc(sizeof(double*) * N_cols);

  _U_dx = (double**)malloc(sizeof(double*) * N_cols);
  _A_x = (double**)malloc(sizeof(double*) * NNZ);
  _B_x = (double**)malloc(sizeof(double*) * N_cols);
  _A_ss = (double**)malloc(sizeof(double*) * N_cols);
  _A_upper = (double**)malloc(sizeof(double*) * NNZ);

  double* _UD_data = Workspace;
  double* _B_data = &Workspace[N_cols * stride];
  const double* _U_d0 = U_ptr + stride * col_offset;
  int64_t* _F_d = (int64_t*)malloc(sizeof(int64_t) * N_cols * R_dim);
  int64_t _F_len = 0;

  for (int64_t x = 0; x < N_cols; x++) {
    int64_t diag_id = 0;
    for (int64_t yx = col_A[x]; yx < col_A[x + 1]; yx++) {
      int64_t y = row_A[yx];
      if (x + col_offset == y)
        diag_id = yx;
      _U_r[yx] = U_ptr + stride * y;
      _U_s[yx] = U_ptr + stride * y + R_dim * N_dim;
      _V_x[yx] = _UD_data + stride * x;
      _A_x[yx] = A_ptr + stride * yx;
      _A_upper[yx] = A_up[yx];
    }

    _A_d[x] = A_ptr + stride * diag_id;
    _B_x[x] = _B_data + stride * x;
    _U_d[x] = U_ptr + stride * (x + col_offset);
    _A_rs[x] = A_ptr + stride * diag_id + R_dim;
    _U_dx[x] = _UD_data + stride * x;
    _A_ss[x] = A_up[diag_id];
    _A_sx[x] = _B_data + stride * x + R_dim * N_dim;

    int64_t dimc = dimr[x + col_offset];
    int64_t fill_new = R_dim - dimc;
    for (int64_t i = 0; i < fill_new; i++)
      _F_d[_F_len + i] = x * stride + (N_dim + 1) * (dimc + i);
    _F_len = _F_len + fill_new;
  }
  
  struct BatchedFactorParams* params_ptr = (struct BatchedFactorParams*)malloc(sizeof(struct BatchedFactorParams));
  params_ptr->N_r = R_dim;
  params_ptr->N_s = S_dim;
  params_ptr->N_upper = N_up;
  params_ptr->L_diag = N_cols;
  params_ptr->L_nnz = NNZ;
  params_ptr->L_fill = _F_len;
  hipMalloc((void**)&(params_ptr->F_d), sizeof(int64_t) * N_cols * R_dim);

  hipMalloc((void**)&(params_ptr->A_d), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->U_d), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->U_r), sizeof(double*) * NNZ);
  hipMalloc((void**)&(params_ptr->U_s), sizeof(double*) * NNZ);
  hipMalloc((void**)&(params_ptr->V_x), sizeof(double*) * NNZ);
  hipMalloc((void**)&(params_ptr->A_rs), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->A_sx), sizeof(double*) * N_cols);

  hipMalloc((void**)&(params_ptr->U_dx), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->A_x), sizeof(double*) * NNZ);
  hipMalloc((void**)&(params_ptr->B_x), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->A_ss), sizeof(double*) * N_cols);
  hipMalloc((void**)&(params_ptr->A_upper), sizeof(double*) * NNZ);

  params_ptr->U_d0 = _U_d0;
  params_ptr->UD_data = _UD_data;
  params_ptr->B_data = _B_data;

  hipMalloc((void**)&(params_ptr->info), sizeof(int) * N_cols);
  *params = params_ptr;

  hipMemcpy(params_ptr->F_d, _F_d, sizeof(int64_t) * N_cols * R_dim, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_d, _A_d, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->U_d, _U_d, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->U_r, _U_r, sizeof(double*) * NNZ, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->U_s, _U_s, sizeof(double*) * NNZ, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->V_x, _V_x, sizeof(double*) * NNZ, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_rs, _A_rs, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_sx, _A_sx, sizeof(double*) * N_cols, hipMemcpyHostToDevice);

  hipMemcpy(params_ptr->U_dx, _U_dx, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_x, _A_x, sizeof(double*) * NNZ, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->B_x, _B_x, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_ss, _A_ss, sizeof(double*) * N_cols, hipMemcpyHostToDevice);
  hipMemcpy(params_ptr->A_upper, _A_upper, sizeof(double*) * NNZ, hipMemcpyHostToDevice);

  free(_F_d);
  free(_A_d);
  free(_U_d);
  free(_U_r);
  free(_U_s);
  free(_V_x);
  free(_A_rs);
  free(_A_sx);
  free(_U_dx);
  free(_A_x);
  free(_B_x);
  free(_A_ss);
  free(_A_upper);
}

void batchParamsDestory(void* params) {
  struct BatchedFactorParams* params_ptr = (struct BatchedFactorParams*)params;
  if (params_ptr->F_d)
    hipFree(params_ptr->F_d);
  if (params_ptr->A_d)
    hipFree(params_ptr->A_d);
  if (params_ptr->U_d)
    hipFree(params_ptr->U_d);
  if (params_ptr->U_r)
    hipFree(params_ptr->U_r);
  if (params_ptr->U_s)
    hipFree(params_ptr->U_s);
  if (params_ptr->V_x)
    hipFree(params_ptr->V_x);
  if (params_ptr->A_rs)
    hipFree(params_ptr->A_rs);
  if (params_ptr->A_sx)
    hipFree(params_ptr->A_sx);
  if (params_ptr->U_dx)
    hipFree(params_ptr->U_dx);
  if (params_ptr->A_x)
    hipFree(params_ptr->A_x);
  if (params_ptr->B_x)
    hipFree(params_ptr->B_x);
  if (params_ptr->A_ss)
    hipFree(params_ptr->A_ss);
  if (params_ptr->A_upper)
    hipFree(params_ptr->A_upper);
  if (params_ptr->info)
    hipFree(params_ptr->info);

  free(params);
}

void batchCholeskyFactor(void* params_ptr) {
  struct BatchedFactorParams* params = (struct BatchedFactorParams*)params_ptr;
  int64_t U = params->N_upper, R = params->N_r, S = params->N_s, N = R + S, D = params->L_diag;
  double one = 1., zero = 0., minus_one = -1.;

  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, R, N, &one, 
    params->A_d, N, params->U_d, N, &zero, params->U_dx, N, D);
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, R, R, N, &one, 
    params->U_d, N, (const double**)(params->U_dx), N, &zero, params->B_x, N, D);
  hipblasDcopy(cublasH, N * N * D, params->U_d0, 1, params->UD_data, 1);
  thrust::fill(thrust::cuda::par.on(stream), thrust::make_permutation_iterator(params->B_data, params->F_d), 
    thrust::make_permutation_iterator(params->B_data, params->F_d + params->L_fill), 1.);

  hipsolverDnDpotrfBatched(cusolverH, HIPBLAS_FILL_MODE_LOWER, R, params->B_x, N, params->info, D);
  hipblasDtrsmBatched(cublasH, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, 
    N, R, &one, (const double**)(params->B_x), N, params->U_dx, N, D);

  for (int64_t i = 0; i < params->L_nnz; i += D) {
    int64_t len = params->L_nnz - i > D ? D : params->L_nnz - i;
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &one, 
      (const double**)(&params->A_x[i]), N, &params->V_x[i], N, &zero, params->B_x, N, len);
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, N, R, N, &one, 
      &params->U_r[i], N, (const double**)(params->B_x), N, &zero, &params->A_x[i], N, len);
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, S, S, N, &one, 
      &params->U_s[i], N, params->A_sx, N, &zero, &params->A_upper[i], U, len);
  }
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, S, S, R, &minus_one, 
    params->A_rs, N, params->A_rs, N, &one, params->A_ss, U, D);
  
  hipStreamSynchronize(stream);
}

void chol_decomp(double* A, int64_t Nblocks, int64_t block_dim, const int64_t dims[]) {
  int64_t lda = Nblocks * block_dim;
  int64_t row = 0;
  for (int64_t i = 0; i < Nblocks; i++) {
    int64_t Arow = i * block_dim;
    if (row < Arow)
      for (int64_t j = 0; j < dims[i]; j++) {
        int64_t rj = row + j;
        int64_t arj = Arow + j;
        hipblasDswap(cublasH, lda - rj, &A[rj * (lda + 1)], 1, &A[arj * lda + rj], 1);
        hipblasDswap(cublasH, rj + 1, &A[rj], lda, &A[arj], lda);
      }
    row = row + dims[i];
  }

  int* info, Lwork;
  double* Workspace;
  hipsolverDnDpotrf_bufferSize(cusolverH, HIPBLAS_FILL_MODE_LOWER, row, A, lda, &Lwork);
  hipMalloc((void**)&Workspace, sizeof(double) * Lwork);
  hipMalloc((void**)&info, sizeof(int));
  hipsolverDnDpotrf(cusolverH, HIPBLAS_FILL_MODE_LOWER, row, A, lda, Workspace, Lwork, info);
  hipStreamSynchronize(stream);
  hipFree(Workspace);
  hipFree(info);
}
